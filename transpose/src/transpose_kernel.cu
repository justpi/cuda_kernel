#include "hip/hip_runtime.h"
#include "transpose_kernel.h"
#include <hip/hip_runtime.h>


__global__ void transpose_baseline(float* d_a, float * d_b, int M, int N) {
    /* base版本的transpose函数 */
    int idx_x = blockDim.x * blockIdx.x + threadIdx.x;
    int idx_y = blockDim.x * blockIdx.x + threadIdx.x;
    d_b[idx_x * M + idx_y] = d_a[idx_y * N + idx_x];
}



void transpose_kernel_launcher(float* h_a, float* h_b, int M, int N) {
    /* transpose核函数launcher:
        h_a: host端数组
        h_b: host端输出
        N: 数组a的col
        M：数组a的row
     */
    int size = N * M;
    /* 分配global mem空间 */
    float* d_a;
    float* d_b;

    CUDA_CHECK(hipMalloc(&d_a, size * sizeof(float)));
    CUDA_CHECK(hipMalloc(&d_b, size * sizeof(float)));

    /* 数据拷贝 */
    CUDA_CHECK(hipMemcpy(d_a, h_a, size * sizeof(float), hipMemcpyHostToDevice));

    /* 核函数执行 */
    dim3 block(BLOCK_SIZE, BLOCK_SIZE);
    dim3 grid((M + BLOCK_SIZE - 1) / BLOCK_SIZE, (M + BLOCK_SIZE - 1) / BLOCK_SIZE);
    
    transpose_baseline<<<grid, block>>>(d_a, d_b, M, N);

    /* 数据拷回 */
    CUDA_CHECK(hipMemcpy(h_b, d_b, size * sizeof(float), hipMemcpyDeviceToHost));
    CUDA_CHECK(hipDeviceSynchronize());

    /* Free */
    CUDA_CHECK(hipFree(d_b));
    CUDA_CHECK(hipFree(d_a));
}