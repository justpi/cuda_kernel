#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#define FETCH_FLOAT4(pointer) (reinterpret_cast<float4*>(&(pointer))[0]) 
/*1. element-wise op*/

/*1.1 add */

// block(256)：每个线程计算一个element
// grid(round(N/256))
// a: N, b: N, out:N
__global__ void add_base(float* a, float* b, float* out) {
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    out[idx] = a[idx] * b[idx];
}

// block(256)   每个线程计算4个elements
// grid(round(N/(256*4)))
// a: N, b: N, out:N
__global__ void add_vec4(float* a, float* b, float* out) {
    const int idx = (blockIdx.x * blockDim.x + threadIdx.x) * 4;
    float4 reg_a = FETCH_FLOAT4(a[idx]);
    float4 reg_b = FETCH_FLOAT4(b[idx]);
    float4 reg_out;
    reg_out.x = reg_a.x + reg_b.x;
    reg_out.y = reg_a.y + reg_b.y;
    reg_out.z = reg_a.z + reg_b.z;
    reg_out.w = reg_a.w + reg_b.w;
    FETCH_FLOAT4(out[idx]) = reg_out;
}


/*1.2 dot product */

// block(256): 每个线程计算一个element
// grid([N/256])
// a: N, b: N, out: N
__global__ void dot_base(float* a, float* b, float* out) {
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    out[idx] = a[idx] * b[idx];
}

// block(256): 每个线程计算4个人element
// grid([N/(256 * 4)])
// a:N, b:N, out:N
__global__ void dot_vec4(float* a, float* b, float* out) {
    const int idx = (blockIdx.x * blockDim.x + threadIdx.x) * 4;
    float4 reg_a = FETCH_FLOAT4(a[idx]);
    float4 reg_b = FETCH_FLOAT4(b[idx]);
    float4 reg_out;
    reg_out.x = reg_a.x * reg_b.x;
    reg_out.y = reg_a.y * reg_b.y;
    reg_out.z = reg_a.z * reg_b.z;
    reg_out.w = reg_a.w * reg_b.w;
    FETCH_FLOAT4(out[idx]) = reg_out;
}

/*1.3 dot product pro: 先求点积再求和，相较1.2多了求和的过程*/

// 每个warp计算的reduce sum，使用__shfl_xor_sync原语进行一个warp内的线程通信
template <const int WarpSize=32>
__device__ int warp_reduce_add(float val) {
    #pragma unroll
    for (int mask=WarpSize>>1; mask >= 1; mask>>=1) {
        val += __shfl_xor_sync(0xffffffff, val, mask);
    }
    return val;
}

// 每个block的reduce sum计算方法，使用多个warp进行block的计算
template <const int NUM_THREADS,
        const int WarpSize
        >
__device__ block_reduce_add(float val) {

    
    const int tid = threadIdx.x;
    const int idx = blockIdx.x * blockDim.x + tid;
    const int warp_idx = tid / WarpSize;
    const int lane_idx = tid % WarpSize;

    const int NUM_WARPS = (NUM_THREADS + WarpSize - 1) / WarpSize;
    __shared__ float sdata[NUM_WAPRS];
    val = warp_reduce_add(val);
    if(lane == 0) sdata[warp_idx] = val;

    val = (lane < NUM_WAPRS) ? sdata[lane]:0.;
    val = warp_reduce_add(val);
    return val;
}

// block(256)
// grid([N/256])
// a: Nx1, b:Nx1, out:1
template <const int N>
__global__ void dot_product_base(float* a, float* b, float* out) {
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    float val = a[idx] * b[idx];
    float output = block_reduce_add(val);
    if (tid == 0) out[0] = output;
}

// block(256)
// grid([N/ 256*4])
// a: Nx1, b: Nx1, out:1
template <const int N> 
__global__ void dot_product_vec4(float* a, float* b, float *out) {
    const int idx = (blockIdx.x * blockDim.x + threadIdx.x) * 4;
    float4 reg_out;
    float4 reg_a = FETCH_FLOAT4(a[idx]);
    float4 reg_b = FETCH_FLOAT4(b[idx]);
    reg_out.x = reg_a.x * reg_b.x;
    reg_out.y = reg_a.y * reg_b.y;
    reg_out.z = reg_a.z * reg_b.z;
    reg_out.w = reg_a.w * reg_b.w;
    float val = reg.x + reg_out.y + reg_out.z + reg_out.w;
    float output = block_reduce_add(val);
    if (tid == 0) out[0] = output;
}

/*1.4 histgram*/

// block(256)
// grid([N/256])
// a: Nx1, out: counted hisgram
__global__ void histgram_base(int* a, int* out, int N) {
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N) atomicAdd(&out[a[idx]], 1);
}


// block(256)
// grid([N / (256*4)])
// a: Nx1, out: counted histgram
__global__ void histgram_vec4(int* a, int* out, int N) {
    const int idx = (blockIdx.x * blockDim.x + threadIdx.x) * 4;
    if (idx < N) {
        float4 reg_a = FETCH_FLOAT4(a[idx]);
        atomicAdd(&out[reg_a.x], 1);
        atomicAdd(&out[reg_a.y], 1);
        atomicAdd(&out[reg_a.z], 1);
        atomicAdd(&out[reg_a.w], 1)
    }
}


/*1.5 sigmoid*/

__device__ __forceinline__ float sigmoid_function(float x) {
    return (1.0f / expf(-x) + 1.0f);
}
// block(256)
// grid([N/256])
// x: Nx1, y: Nx1
__global__ void sigmoid_base(float *x, float *y, int N) {
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N) {
        y[idx] = sigmoid_function(x[idx]);
    }
}

// block(256 / 4)
// grid([N/256])
// x: Nx1, y: Nx1
__global__ void sigmoid_vec4(float *x, float *y, int N) {
    const int idx = (blockIdx.x * blockDim.x + threadIdx.x) * 4;
    if (idx < N) {
        float4 reg_x = FETCH_FLOAT4(x[idx]);
        FETCH_FLOAT4(y[idx]) = {
            sigmoid_function(reg_x.x),
            sigmoid_function(reg_x.y),
            sigmoid_function(reg_x.z),
            sigmoid_function(reg_x.w)
        };
    }
}

/*1.6 relu*/

__device__ __forceinline__ float relu_function(float x) {
    return fmaxf(0.0f, x);
}

__device__ forceinline__ float leaky_relu_function(float x) {
    return fmaxf(0.1f * x, x);
}

// block(256)
// grid([N/256])
// x: Nx1, y: Nx1
__global__ void relu_base(float* x, float* y, int N) {
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N) {
        y[idx] = relu_function(x[idx]);
    }
}

// block(256/4)
// grid([N/256])
// x: Nx1, y: Nx1
__global__ void relu_vec4(float* x, float* y, int N) {
    const int idx = (blockIdx.x * blockDim.x + threadIdx.x);
    if (idx < N) {
        float4 reg_x = FETCH_FLOAT4(x[idx]);
        FETCH_FLOAT4(y[idx]) = {
            relu_function(reg_x.x),
            relu_function(reg_x.y),
            relu_function(reg_x.z),
            relu_function(reg_x.w),
        };
    } 
}



/* 2.transpose */
#define OFFSET(row, col, ld) ((row) * (ld) + (col))
// block(32， 32)
// grid(M/32, N/32)
// x: MxN, y: NxM
__global__ void transpose_base(float* a, float* b, int M, int N) {
    const int idx_x = blockIdx.x * blockDim.x + threadIdx.x;
    const int idx_y = blockIdx.y * blockDim.y + threadIdx.y;
    if (idx_x < M)
    b[OFFSET(idx_x, idx_y, M)] = a[OFFSET(idx_y, idx_x, N)];
}

// block(32, 32) 
// grid(M/32, N/32)
// x: MxN, y: NxM
template<int BLOCK_SIZE>
__global__ void transpose_shared(float* a, float* b, int M, int N) {    // 运行时传入M, N，适用与M, N未知的情况
    const int idx_x = blockIdx.x * blockDim.x + threadIdx.x;
    const int idx_y = blockIdx.y * blockDim.y + threadIdx.y;
    __shared__ float sdata[BLOCK_SIZE][BLOCK_SIZE];
    sdata[threadIdx.y][threadIdx.x] = a[OFFSET(idx_y, idx_x, N)];
    __syncthreads();
    b[OFFSET(idx_x, idx_y, M)] = sdata[threadIdx.x][threadIdx.y];
}


/*3. reduce*/

