#include "hip/hip_runtime.h"
#include "conv_kernel.h"
#include <hip/hip_runtime.h>
#include <hipDNN.h>
#include <hipblas.h>

#include "cutlass/conv/device/implicit_gemm_convolution.h"
#include "cutlass/conv/kernel/default_conv2d_fprop.h"
#include "cutlass/gemm/device/gemm.h"
#include "cutlass/cutlass.h"
#include "cutlass/util/host_tensor.h"
#include "cutlass/util/reference/host/tensor_fill.h"

#define div_ceil(a, b) ((a + b - 1) / b)

#define FETCH_FLOAT4(pointer) (reinterpret_cast<float4*>(&(pointer))[0])

void conv_cudnn_launcher(float* input, float* output, float* weight, 
                        int batch, int in_channel, int out_channel, int height, int width, 
                        int kheight, int kwidth, int pad_height, int pad_width, int stride_height, int stride_width) {
    

    hipdnnHandle_t cudnn;
    hipdnnCreate(&cudnn);
    /*设置输入张量参数*/
    hipdnnTensorDescriptor_t input_desc;
    hipdnnCreateTensorDescriptor(&input_desc);
    hipdnnSetTensor4dDescriptor(input_desc,
                            /*format=*/HIPDNN_TENSOR_NCHW,
                            /*dataType=*/HIPDNN_DATA_FLOAT,
                            /*batch_size=*/batch,
                            /*channel=*/in_channel,
                            /*height=*/height,
                            /*width=*/width
                        );
     
    /*设置过滤器参数*/
    hipdnnFilterDescriptor_t filter_desc;
    hipdnnCreateFilterDescriptor(&filter_desc);
    hipdnnSetFilter4dDescriptor(filter_desc,
                            HIPDNN_DATA_FLOAT,
                            HIPDNN_TENSOR_NCHW,
                            out_channel,
                            in_channel,
                            kheight,
                            kwidth
    );

    /*设置卷积算子参数*/
    hipdnnConvolutionDescriptor_t conv_desc;
    hipdnnCreateConvolutionDescriptor(&conv_desc);
    hipdnnSetConvolution2dDescriptor(conv_desc,
                                /*pad_height=*/pad_height,
                                /*pad_width=*/pad_width,
                                /*stride_height=*/stride_height,
                                /*stride_width=*/stride_width,
                                /*dilation_height=*/1,
                                /*dilation_width=*/1,
                                /*mode=*/HIPDNN_CROSS_CORRELATION,
                                /*computeType=*/HIPDNN_DATA_FLOAT

    );

    /*输出张量设置*/
    int b_out, c_out, h_out, w_out;
    hipdnnGetConvolution2dForwardOutputDim(conv_desc,
                                        input_desc,
                                        filter_desc,
                                        &b_out,
                                        &c_out,
                                        &h_out,
                                        &w_out

    );
    hipdnnTensorDescriptor_t output_desc;
    hipdnnCreateTensorDescriptor(&output_desc);
    hipdnnSetTensor4dDescriptor(output_desc,
                            HIPDNN_TENSOR_NCHW,
                            HIPDNN_DATA_FLOAT,
                            b_out,
                            c_out,
                            h_out,
                            w_out

    );

    /*分配显存*/
    int input_size = batch * in_channel * height * width;
    int weight_size = out_channel * in_channel * kheight * kwidth;
    int output_size = b_out * c_out * h_out * w_out;

    float *input_d, *weight_d, *output_d;
    CUDA_CHECK(hipMalloc(&input_d, input_size * sizeof(float)));
    CUDA_CHECK(hipMalloc(&weight_d, weight_size * sizeof(float)));
    CUDA_CHECK(hipMalloc(&output_d, output_size * sizeof(float)));

    CUDA_CHECK(hipMemcpy(input_d, input, input_size * sizeof(float), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(weight_d, weight, weight_size * sizeof(float), hipMemcpyHostToDevice));

    /*执行计算*/
    float alpha = 1.0f, beta = 0.0f;
    hipdnnConvolutionForward(cudnn,
                        &alpha,
                        input_desc,
                        input_d,
                        filter_desc,
                        weight_d,
                        conv_desc,
                        HIPDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_GEMM,
                        NULL,
                        0,
                        &beta,
                        output_desc,
                        output_d

    );

    /*数据拷回*/
    CUDA_CHECK(hipMemcpy(output, output_d,output_size * sizeof(float), hipMemcpyDeviceToHost));
    CUDA_CHECK(hipDeviceSynchronize());
    LAST_KERNEL_CHECK();

    /*清除空间*/
    hipFree(output_d);
    hipFree(weight_d);
    hipFree(input_d);
    hipdnnDestroyTensorDescriptor(output_desc);
    hipdnnDestroyTensorDescriptor(input_desc);
    hipdnnDestroyFilterDescriptor(filter_desc);
    hipdnnDestroyConvolutionDescriptor(conv_desc);
    hipdnnDestroy(cudnn);
    
}

__global__ void conv_implicit_gemm_base(float* input, float* output, float* weight, 
    int batch, int in_channel, int out_channel, int height, int width, int height_out, int width_out,
    int kheight, int kwidth, int pad_height, int pad_width, int stride_h, int stride_w) {
    /*首先计算当前线程的输出索引，然后计算weight的im2col索引，从而计算出weight的各项参数，最后计算input的各项参数*/
    __shared__ float sdata_i[TILE][TILE+1];
    __shared__ float sdata_w[TILE][TILE+1];
    const int out_ch = blockIdx.y * TILE + threadIdx.y;
    const int out_hw = blockIdx.x * TILE + threadIdx.x;
    const int b = blockIdx.z;
    const int ty = threadIdx.y;
    const int tx = threadIdx.x;
    const int out_w = out_hw % height_out;
    const int out_h = out_hw / height_out;
    /*输出纬度为K*PQ，矩阵A为K*CRS，，im2col_weight的形状是K*CRS，表示矩阵A的M和K，矩阵B为CRS*PQ，计算CRS和PQ，即为im2col_input的K和N*/
    // const int weight_im2col_M = out_channel;
    const int weight_im2col_K = in_channel * kheight * kwidth;
    // const int input_im2col_N = height_out * width_out;
    const int k_tile = div_ceil(weight_im2col_K, TILE);
    int in_ch, kh, kw, in_h, in_w;
    float output_value = 0.0f;
    for (int i=0; i < k_tile; ++i) {
        /*计算weight的索引*/
        int idx_k = i * TILE + ty;
        in_ch = idx_k / (kheight * kwidth);
        kh = (idx_k % (kheight * kwidth)) / kwidth;
        kw = idx_k % kwidth;
        /*计算input索引*/
        in_h = out_h * stride_h + kh - pad_height;
        in_w = out_w * stride_w + kw - pad_width;
        /*将weight矩阵读取到shared memory中*/
        if (i * TILE + tx >= weight_im2col_K) sdata_w[ty][tx] = 0.0f;
        else {
            sdata_w[ty][tx] = weight[out_ch * weight_im2col_K + i * TILE + tx];
        }
        /*将input矩阵读取到shared memory中*/
        if (in_h >=0 && in_h < height && in_w >= 0 && in_w < width) {
            sdata_i[ty][tx] = input[b * in_channel * height * width + in_ch * height * width + in_h * width + in_w];
        }
        else sdata_i[ty][tx] = 0.0f;
        __syncthreads();
        /*计算当前tile的输出值*/
        for (int k=0; k < TILE; ++k) {
            output_value += sdata_w[ty][k] * sdata_i[k][tx];
        }
        __syncthreads();

    }
    if (out_ch < out_channel && out_hw < height_out * width_out)
    output[b * out_channel * height_out * width_out + out_ch * height_out * width_out + out_h * width_out + out_w] = output_value;

}


__global__ void conv_implicit_gemm_unroll(float* input, float* output, float* weight, 
    int batch, int in_channel, int out_channel, int height, int width, int height_out, int width_out,
    int kheight, int kwidth, int pad_height, int pad_width, int stride_h, int stride_w) {
    /*首先计算当前线程的输出索引，然后计算weight的im2col索引，从而计算出weight的各项参数，最后计算input的各项参数*/
    __shared__ float sdata_i[TILE][TILE];
    __shared__ float sdata_w[TILE][TILE];
    const int out_ch = blockIdx.y * TILE + threadIdx.y;
    const int out_hw = blockIdx.x * TILE + threadIdx.x;
    const int b = blockIdx.z;
    const int ty = threadIdx.y;
    const int tx = threadIdx.x;
    const int out_w = out_hw % height_out;
    const int out_h = out_hw / height_out;
    /*输出纬度为K*PQ，矩阵A为K*CRS，，im2col_weight的形状是K*CRS，表示矩阵A的M和K，矩阵B为CRS*PQ，计算CRS和PQ，即为im2col_input的K和N*/
    // const int weight_im2col_M = out_channel;
    const int weight_im2col_K = in_channel * kheight * kwidth;
    // const int input_im2col_N = height_out * width_out;
    const int k_tile = div_ceil(weight_im2col_K, TILE);
    int in_ch, kh, kw, in_h, in_w;
    int idx_k;
    float output_value = 0.0f;
    #pragma unroll
    for (int i=0; i < k_tile; ++i) {
        /*计算weight的索引*/
        idx_k = i * TILE + ty;
        in_ch = idx_k / (kheight * kwidth);
        kh = (idx_k % (kheight * kwidth)) / kwidth;
        kw = idx_k % kwidth;
        /*计算input索引*/
        in_h = out_h * stride_h + kh - pad_height;
        in_w = out_w * stride_w + kw - pad_width;
        /*将weight矩阵读取到shared memory中*/
        if (i * TILE + tx >= weight_im2col_K) sdata_w[ty][tx] = 0.0f;
        else {
            sdata_w[ty][tx] = weight[out_ch * weight_im2col_K + i * TILE + tx];
        }
        /*将input矩阵读取到shared memory中*/
        if (in_h >=0 && in_h < height && in_w >= 0 && in_w < width) {
            sdata_i[ty][tx] = input[b * in_channel * height * width + in_ch * height * width + in_h * width + in_w];
        }
        else sdata_i[ty][tx] = 0.0f;
        __syncthreads();
        /*计算当前tile的输出值*/
        #pragma unroll
        for (int k=0; k < TILE; ++k) {
            output_value += sdata_w[ty][k] * sdata_i[k][tx];
        }
        __syncthreads();

    }
    if (out_ch < out_channel && out_hw < height_out * width_out)
    output[b * out_channel * height_out * width_out + out_ch * height_out * width_out + out_h * width_out + out_w] = output_value;

}


__global__ void conv_implicit_gemm_vec4(float* input, float* output, float* weight, 
    int batch, int in_channel, int out_channel, int height, int width, int height_out, int width_out,
    int kheight, int kwidth, int pad_height, int pad_width, int stride_h, int stride_w) {
    /*首先计算当前线程的输出索引，然后计算weight的im2col索引，从而计算出weight的各项参数，最后计算input的各项参数*/
    __shared__ float sdata_i[TILE][TILE];
    __shared__ float sdata_w[TILE][TILE];
    const int out_ch = blockIdx.y * TILE + threadIdx.y;
    const int out_hw = blockIdx.x * TILE + threadIdx.x;
    const int b = blockIdx.z;
    const int ty = threadIdx.y;
    const int tx = threadIdx.x;
    const int lane_id = ty * blockDim.x + tx;
    const int out_w = out_hw % height_out;
    const int out_h = out_hw / height_out;
    /*输出纬度为K*PQ，矩阵A为K*CRS，，im2col_weight的形状是K*CRS，表示矩阵A的M和K，矩阵B为CRS*PQ，计算CRS和PQ，即为im2col_input的K和N*/
    // const int weight_im2col_M = out_channel;
    const int weight_im2col_K = in_channel * kheight * kwidth;
    // const int input_im2col_N = height_out * width_out;

    /*使用寄存器存放向量化访存的值*/
    float frag_w[4];
    // float frag_i[4];
    const int k_tile = div_ceil(weight_im2col_K, TILE);
    int in_ch, kh, kw, in_h, in_w;
    float output_value = 0.0f;
    #pragma unroll
    for (int i=0; i < k_tile; ++i) {
        /*计算weight的索引*/
        int idx_k = i * TILE + ty;  // input的row索引
        in_ch = idx_k / (kheight * kwidth);
        kh = (idx_k % (kheight * kwidth)) / kwidth;
        kw = idx_k % kwidth;
        /*计算input索引*/
        in_h = out_h * stride_h + kh - pad_height;
        in_w = out_w * stride_w + kw - pad_width;
        if (lane_id %4 == 0) {
            /*将weight矩阵读取到shared memory中*/
            if (i * TILE + tx >= weight_im2col_K) {
                sdata_w[ty][tx] = 0.0f;
                sdata_w[ty][tx+1] = 0.0f;
                sdata_w[ty][tx+2] = 0.0f;
                sdata_w[ty][tx+3] = 0.0f;

            }
            else {
                FETCH_FLOAT4(frag_w) = FETCH_FLOAT4(weight[out_ch * weight_im2col_K + i * TILE + tx]);
                sdata_w[ty][tx] = frag_w[0];
                sdata_w[ty][tx+1] = frag_w[1];
                sdata_w[ty][tx+2] = frag_w[2];
                sdata_w[ty][tx+3] = frag_w[3];
            }
        }
        /*将input矩阵读取到shared memory中*/
        if (in_h >=0 && in_h < height && in_w >= 0 && in_w < width) {
            sdata_i[ty][tx] = input[b * in_channel * height * width + in_ch * height * width + in_h * width + in_w];
        }
        else sdata_i[ty][tx] = 0.0f;
        __syncthreads();
        
        /*计算当前tile的输出值*/
        #pragma unroll
        for (int k=0; k < TILE; ++k) {
            output_value += sdata_w[ty][k] * sdata_i[k][tx];
        }
        __syncthreads();

    }
    if (out_ch < out_channel && out_hw < height_out * width_out)
    output[b * out_channel * height_out * width_out + out_ch * height_out * width_out + out_h * width_out + out_w] = output_value;

}

__global__ void convert_float_to_cutlass_half(float* input, cutlass::half_t* output, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        cutlass::NumericConverter<cutlass::half_t, float> converter;
        output[idx] = converter(input[idx]);
    }
}

void cutlass_conv(float* input_d, float* output_d, float* weight_d, 
                    int batch, int in_channel, int out_channel, int height, int width, 
                    int height_out, int width_out, 
                    int kheight, int kwidth, 
                    int pad_height, int pad_width, 
                    int stride_height, int stride_width) {

    using ElementInput = cutlass::half_t;
    using ElementOutput = float;
    using ElementWeight = cutlass::half_t;

    using ElementAccumulator = float;

    using LayoutInput = cutlass::layout::TensorNCHW;
    using LayoutOutput = cutlass::layout::TensorNCHW;
    using LayoutWeight = cutlass::layout::TensorNCHW;

    using MMAOp = cutlass::arch::OpClassTensorOp;

    using SmArch = cutlass::arch::Sm80;

    using ThreadblockShape = cutlass::gemm::GemmShape<128, 128, 32>;
    using WarpShape = cutlass::gemm::GemmShape<64, 64, 32>;
    using InstructionShape = cutlass::gemm::GemmShape<16, 8, 16>;

    using SwizzleThreadBlock = cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<>;

    constexpr int NumStages = 3;

    static cutlass::conv::IteratorAlgorithm const IteratorAlgorithm = cutlass::conv::IteratorAlgorithm::kAnalytic;

    static cutlass::conv::StrideSupport const OutputStride = cutlass::conv::StrideSupport::kUnity;

    using EpilogueOp = cutlass::epilogue::thread::LinearCombination<
                        ElementOutput,
                        32 / cutlass::sizeof_bits<ElementOutput>::value,
                        ElementAccumulator,
                        float>;
    
    using Conv2dFpropKernel = typename cutlass::conv::kernel::DefaultConv2dFprop<
                        ElementInput, LayoutInput,
                        ElementWeight, LayoutWeight,
                        ElementOutput,  LayoutOutput,
                        ElementAccumulator,
                        MMAOp,
                        SmArch,
                        ThreadblockShape,
                        WarpShape,
                        InstructionShape,
                        EpilogueOp,
                        SwizzleThreadBlock,
                        NumStages,
                        cutlass::arch::OpMultiplyAdd,
                        IteratorAlgorithm,
                        OutputStride
    >::Kernel;

    using ImplicitGemm = cutlass::conv::device::ImplicitGemmConvolution<Conv2dFpropKernel>;
    
    float alpha = 1.0, beta = 0.0;
    
    /*float2half_t*/
    
    int weight_size = out_channel*in_channel*kheight*kwidth;
    int input_size = batch*in_channel*height*width;
    /*不能保证内存对齐*/
    // cutlass::half_t *input_half, *weight_half;
    // int input_size = batch*in_channel*height*width;
    // CUDA_CHECK(hipMalloc(&input_half, input_size * sizeof(cutlass::half_t)));
    // CUDA_CHECK(hipMalloc(&weight_half, weight_size * sizeof(cutlass::half_t)));

    // dim3 block(BLOCK_SIZE);
    // dim3 grid_input(input_size / BLOCK_SIZE);
    // dim3 grid_weight(weight_size / BLOCK_SIZE);
    // convert_float_to_cutlass_half<<<grid_input, block>>>(input_d, input_half, input_size);
    // convert_float_to_cutlass_half<<<grid_weight, block>>>(weight_d, weight_half, weight_size);

    // 使用对齐的内存分配
    // 使用 CUTLASS 的内存分配工具
    cutlass::HostTensor<ElementInput, LayoutInput> input_half({batch, in_channel, height, width});
    cutlass::HostTensor<ElementWeight, LayoutWeight> weight_half({out_channel, in_channel, kheight, kwidth});

    dim3 block(BLOCK_SIZE);
    dim3 grid_input((input_size + block.x - 1) / block.x);
    dim3 grid_weight((weight_size + block.x - 1) / block.x);
    
    convert_float_to_cutlass_half<<<grid_input, block>>>(input_d, input_half.device_data(), input_size);
    convert_float_to_cutlass_half<<<grid_weight, block>>>(weight_d, weight_half.device_data(), weight_size);
    
    // 确保转换完成
    hipDeviceSynchronize();
        
    cutlass::conv::Mode mode = cutlass::conv::Mode::kCrossCorrelation;

    cutlass::conv::Conv2dProblemSize problem_size(
        {batch, in_channel, height, width},    // input size (NCHW)
        {out_channel, in_channel, kheight, kwidth}, // filter size (KCRS)
        {pad_height, pad_height, pad_width, pad_width}, // padding (pad_h, pad_h, pad_w, pad_w)
        {stride_height, stride_width},          // strides (stride_h, stride_w)
        {1, 1},                                // dilation (dilation_h, dilation_w)
        {batch, out_channel, height_out, width_out},   // output size (NCHW)
        mode,
        1   // split k factor
    );

    typename ImplicitGemm::Arguments arguments{
        problem_size,
        {input_half.device_ref(), LayoutInput::packed({batch, in_channel, height, width})},
        {weight_half.device_ref(), LayoutWeight::packed({out_channel, in_channel, kheight, kwidth})},
        {output_d, LayoutOutput::packed({batch, out_channel, height_out, width_out})},
        {output_d, LayoutOutput::packed({batch, out_channel, height_out, width_out})},
        {alpha, beta}
    };

    ImplicitGemm conv_op;
    
    conv_op(arguments);

}


void conv_kernel_launcher(float* input, float* output, float* weight, 
    int batch, int in_channel, int out_channel, int height, int width, 
    int kheight, int kwidth, int pad_height, int pad_width, int stride_height, int stride_width) {
    /*分配显存*/
    int height_out = (height + 2 * pad_height - (kheight - 1) - 1) / stride_height + 1;
    int width_out = (width + 2 * pad_width - (kwidth - 1) - 1) / stride_width + 1;

    int size_input = batch * in_channel * height * width;
    int size_output = batch * out_channel * height_out * width_out;
    int size_weight = out_channel * in_channel * kheight * kwidth;

    float *input_d, *weight_d, *output_d;
    CUDA_CHECK(hipMalloc(&input_d, size_input * sizeof(float)));
    CUDA_CHECK(hipMalloc(&weight_d, size_weight * sizeof(float)));
    CUDA_CHECK(hipMalloc(&output_d, size_output * sizeof(float)));

    CUDA_CHECK(hipMemcpy(input_d, input, size_input * sizeof(float), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(weight_d, weight, size_weight * sizeof(float), hipMemcpyHostToDevice));

    /*baseline*/

    dim3 block_base(TILE, TILE);
    dim3 grid_base(div_ceil(height_out * width_out, TILE), div_ceil(out_channel, TILE), batch);
    conv_implicit_gemm_base<<<grid_base, block_base>>>(input_d, output_d, weight_d, batch, in_channel, out_channel, height, width, height_out, width_out, kheight, kwidth, pad_height, pad_width, stride_height, stride_width);

    /*展开循环*/
    conv_implicit_gemm_unroll<<<grid_base, block_base>>>(input_d, output_d, weight_d, batch, in_channel, out_channel, height, width, height_out, width_out, kheight, kwidth, pad_height, pad_width, stride_height, stride_width);

    /*向量化访存*/
    conv_implicit_gemm_vec4<<<grid_base, block_base>>>(input_d, output_d, weight_d, batch, in_channel, out_channel, height, width, height_out, width_out, kheight, kwidth, pad_height, pad_width, stride_height, stride_width);

    /*cutlass 实现*/
    cutlass_conv(input_d, output_d, weight_d, batch, in_channel, out_channel, height, width, height_out, width_out, kheight, kwidth, pad_height, pad_width, stride_height, stride_width);
    CUDA_CHECK(hipMemcpy(output, output_d, size_output * sizeof(float), hipMemcpyDeviceToHost));
    CUDA_CHECK(hipDeviceSynchronize());

    CUDA_CHECK(hipFree(output_d));
    CUDA_CHECK(hipFree(weight_d));
    CUDA_CHECK(hipFree(input_d));

}