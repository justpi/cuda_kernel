#include "conv_kernel.h"
#include <hip/hip_runtime.h>
#include <hipDNN.h>
#include <hipblas.h>

void conv_cudnn_launcher(float* input, float* output, float* weight, 
                        int batch, int in_channel, int out_channel, int height, int width, 
                        int kheight, int kwidth, int pad_height, int pad_width, int stride_height, int stride_width) {
    

    hipdnnHandle_t cudnn;
    hipdnnCreate(&cudnn);
    /*设置输入张量参数*/
    hipdnnTensorDescriptor_t input_desc;
    hipdnnCreateTensorDescriptor(&input_desc);
    hipdnnSetTensor4dDescriptor(input_desc,
                            /*format=*/HIPDNN_TENSOR_NCHW,
                            /*dataType=*/HIPDNN_DATA_FLOAT,
                            /*batch_size=*/batch,
                            /*channel=*/in_channel,
                            /*height=*/height,
                            /*width=*/width
                        );
     
    /*设置过滤器参数*/
    hipdnnFilterDescriptor_t filter_desc;
    hipdnnCreateFilterDescriptor(&filter_desc);
    hipdnnSetFilter4dDescriptor(filter_desc,
                            HIPDNN_DATA_FLOAT,
                            HIPDNN_TENSOR_NCHW,
                            out_channel,
                            in_channel,
                            kheight,
                            kwidth
    );

    /*设置卷积算子参数*/
    hipdnnConvolutionDescriptor_t conv_desc;
    hipdnnCreateConvolutionDescriptor(&conv_desc);
    hipdnnSetConvolution2dDescriptor(conv_desc,
                                /*pad_height=*/pad_height,
                                /*pad_width=*/pad_width,
                                /*stride_height=*/stride_height,
                                /*stride_width=*/stride_width,
                                /*dilation_height=*/1,
                                /*dilation_width=*/1,
                                /*mode=*/HIPDNN_CROSS_CORRELATION,
                                /*computeType=*/HIPDNN_DATA_FLOAT

    );

    /*输出张量设置*/
    int b_out, c_out, h_out, w_out;
    hipdnnGetConvolution2dForwardOutputDim(conv_desc,
                                        input_desc,
                                        filter_desc,
                                        &b_out,
                                        &c_out,
                                        &h_out,
                                        &w_out

    );
    hipdnnTensorDescriptor_t output_desc;
    hipdnnCreateTensorDescriptor(&output_desc);
    hipdnnSetTensor4dDescriptor(output_desc,
                            HIPDNN_TENSOR_NCHW,
                            HIPDNN_DATA_FLOAT,
                            b_out,
                            c_out,
                            h_out,
                            w_out

    );

    /*分配显存*/
    int input_size = batch * in_channel * height * width;
    int weight_size = out_channel * kheight * kwidth;
    int output_size = b_out * c_out * h_out * w_out;

    float *input_d, *weight_d, *output_d;
    CUDA_CHECK(hipMalloc(&input_d, input_size * sizeof(float)));
    CUDA_CHECK(hipMalloc(&weight_d, weight_size * sizeof(float)));
    CUDA_CHECK(hipMalloc(&output_d, output_size * sizeof(float)));

    CUDA_CHECK(hipMemcpy(input_d, input, input_size * sizeof(float), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(weight_d, weight, weight_size * sizeof(float), hipMemcpyHostToDevice));

    /*执行计算*/
    float alpha = 1.0f, beta = 0.0f;
    hipdnnConvolutionForward(cudnn,
                        &alpha,
                        input_desc,
                        input_d,
                        filter_desc,
                        weight_d,
                        conv_desc,
                        HIPDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_GEMM,
                        NULL,
                        0,
                        &beta,
                        output_desc,
                        output_d

    );

    /*数据拷回*/
    CUDA_CHECK(hipMemcpy(output, output_d,output_size * sizeof(float), hipMemcpyDeviceToHost));
    CUDA_CHECK(hipDeviceSynchronize());
    LAST_KERNEL_CHECK();

    /*清除空间*/
    hipFree(output_d);
    hipFree(weight_d);
    hipFree(input_d);
    hipdnnDestroyTensorDescriptor(output_desc);
    hipdnnDestroyTensorDescriptor(input_desc);
    hipdnnDestroyFilterDescriptor(filter_desc);
    hipdnnDestroyConvolutionDescriptor(conv_desc);
    hipdnnDestroy(cudnn);
    
}