#include "hip/hip_runtime.h"
#include "conv_kernel.h"
#include <hip/hip_runtime.h>
#include <hipDNN.h>
#include <hipblas.h>

#define div_ceil(a, b) ((a + b - 1) / b)

void conv_cudnn_launcher(float* input, float* output, float* weight, 
                        int batch, int in_channel, int out_channel, int height, int width, 
                        int kheight, int kwidth, int pad_height, int pad_width, int stride_height, int stride_width) {
    

    hipdnnHandle_t cudnn;
    hipdnnCreate(&cudnn);
    /*设置输入张量参数*/
    hipdnnTensorDescriptor_t input_desc;
    hipdnnCreateTensorDescriptor(&input_desc);
    hipdnnSetTensor4dDescriptor(input_desc,
                            /*format=*/HIPDNN_TENSOR_NCHW,
                            /*dataType=*/HIPDNN_DATA_FLOAT,
                            /*batch_size=*/batch,
                            /*channel=*/in_channel,
                            /*height=*/height,
                            /*width=*/width
                        );
     
    /*设置过滤器参数*/
    hipdnnFilterDescriptor_t filter_desc;
    hipdnnCreateFilterDescriptor(&filter_desc);
    hipdnnSetFilter4dDescriptor(filter_desc,
                            HIPDNN_DATA_FLOAT,
                            HIPDNN_TENSOR_NCHW,
                            out_channel,
                            in_channel,
                            kheight,
                            kwidth
    );

    /*设置卷积算子参数*/
    hipdnnConvolutionDescriptor_t conv_desc;
    hipdnnCreateConvolutionDescriptor(&conv_desc);
    hipdnnSetConvolution2dDescriptor(conv_desc,
                                /*pad_height=*/pad_height,
                                /*pad_width=*/pad_width,
                                /*stride_height=*/stride_height,
                                /*stride_width=*/stride_width,
                                /*dilation_height=*/1,
                                /*dilation_width=*/1,
                                /*mode=*/HIPDNN_CROSS_CORRELATION,
                                /*computeType=*/HIPDNN_DATA_FLOAT

    );

    /*输出张量设置*/
    int b_out, c_out, h_out, w_out;
    hipdnnGetConvolution2dForwardOutputDim(conv_desc,
                                        input_desc,
                                        filter_desc,
                                        &b_out,
                                        &c_out,
                                        &h_out,
                                        &w_out

    );
    hipdnnTensorDescriptor_t output_desc;
    hipdnnCreateTensorDescriptor(&output_desc);
    hipdnnSetTensor4dDescriptor(output_desc,
                            HIPDNN_TENSOR_NCHW,
                            HIPDNN_DATA_FLOAT,
                            b_out,
                            c_out,
                            h_out,
                            w_out

    );

    /*分配显存*/
    int input_size = batch * in_channel * height * width;
    int weight_size = out_channel * in_channel * kheight * kwidth;
    int output_size = b_out * c_out * h_out * w_out;

    float *input_d, *weight_d, *output_d;
    CUDA_CHECK(hipMalloc(&input_d, input_size * sizeof(float)));
    CUDA_CHECK(hipMalloc(&weight_d, weight_size * sizeof(float)));
    CUDA_CHECK(hipMalloc(&output_d, output_size * sizeof(float)));

    CUDA_CHECK(hipMemcpy(input_d, input, input_size * sizeof(float), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(weight_d, weight, weight_size * sizeof(float), hipMemcpyHostToDevice));

    /*执行计算*/
    float alpha = 1.0f, beta = 0.0f;
    hipdnnConvolutionForward(cudnn,
                        &alpha,
                        input_desc,
                        input_d,
                        filter_desc,
                        weight_d,
                        conv_desc,
                        HIPDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_GEMM,
                        NULL,
                        0,
                        &beta,
                        output_desc,
                        output_d

    );

    /*数据拷回*/
    CUDA_CHECK(hipMemcpy(output, output_d,output_size * sizeof(float), hipMemcpyDeviceToHost));
    CUDA_CHECK(hipDeviceSynchronize());
    LAST_KERNEL_CHECK();

    /*清除空间*/
    hipFree(output_d);
    hipFree(weight_d);
    hipFree(input_d);
    hipdnnDestroyTensorDescriptor(output_desc);
    hipdnnDestroyTensorDescriptor(input_desc);
    hipdnnDestroyFilterDescriptor(filter_desc);
    hipdnnDestroyConvolutionDescriptor(conv_desc);
    hipdnnDestroy(cudnn);
    
}

__global__ void conv_implicit_gemm_base(float* input, float* output, float* weight, 
    int batch, int in_channel, int out_channel, int height, int width, int height_out, int width_out,
    int kheight, int kwidth, int pad_height, int pad_width, int stride_height, int stride_width) {
    /*首先计算当前线程的输出索引，然后计算weight的im2col索引，从而计算出weight的各项参数，最后计算input的各项参数*/
    __shared__ float sdata_i[TILE][TILE];
    __shared__ float sdata_w[TILE][TILE];
    const int out_ch = blockIdx.y * TILE + threadIdx.y;
    const int out_hw = blockIdx.x * TILE + threadIdx.x;
    const int out_w = out_hw % height_out;
    const int out_h = out_hw / height_out;

}


void conv_kernel_launcher(float* input, float* output, float* weight, 
    int batch, int in_channel, int out_channel, int height, int width, 
    int kheight, int kwidth, int pad_height, int pad_width, int stride_height, int stride_width) {
    /*分配显存*/
    int height_out = (height + 2 * padding_h - (kheight - 1) - 1) / stride_h + 1;
    int width_out = (width + 2 * padding_w - (kwidth - 1) - 1) / stride_w + 1;

    int size_input = batch * in_channel * height * width;
    int size_output = batch * out_channel * height_out * width_out;
    int size_weight = out_channel * in_channel * kheight * kwidth;

    float *input_d, *weight_d, *output_d;
    CUDA_CHECK(hipMalloc(&input_d, size_input * sizeof(float)));
    CUDA_CHECK(hipMalloc(&weight_d, size_weight * sizeof(float)));
    CUDA_CHECK(hipMalloc(&output_d, size_output * sizeof(float)));

    CUDA_CHECK(hipMemcpy(input_d, input, size_input * sizeof(float), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(weight_d, weight, size_weight * sizeof(float), hipMemcpyHostToDevice));

    /*baseline*/

    dim3 block_base(TILE, TILE);
    dim3 grid_base(div_ceil(height_out * width_out, TILE), div_ceil(out_channel, TILE), batch);
    conv_implicit_gemm_base<<<grid_base, block_base>>>(input_d, output_d, weight_d, batch, in_channel, out_channel, height, width, kheight, kwidth, pad_height, pad_width, stride_height, stride_width);

    CUDA_CHECK(hipMemcpy(output, output_d, size_output * sizeof(float), hipMemcpyDeviceToHost));
    CUDA_CHECK(hipDeviceSynchronize());

    CUDA_CHECK(hipFree(output_d));
    CUDA_CHECK(hipFree(weight_d));
    CUDA_CHECK(hipFree(input_d));

}