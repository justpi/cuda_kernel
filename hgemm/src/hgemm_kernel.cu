#include "hip/hip_runtime.h"
#include "hgemm_kernel.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include <hipblas.h>
#include <hip/hip_fp16.h>
#include <mma.h>

using namespace nvcuda;

/* half4读取数据的宏 */
#define OFFSET(row, col, ld) ((row) * (ld) + (col))
// 此宏将传递给它的指针 pointer 解释为指向 half4 类型数据的指针，并提取该位置的 half4 值。
#define FETCH_HALF2(pointer) (reinterpret_cast<half2*>(&(pointer))[0])

#define div_ceil(a, b) ((a + b - 1) / b)


__global__ void gemm_baseline(half* d_a, half* d_b, half* d_c, int N, int K, int M) {
    /* baseline:
        d_a: M * K
        d_b: K * N
        d_c: M * N
    */
    /* 计算d_c索引 (i, j) */
    unsigned int c_i = blockDim.x * blockIdx.x + threadIdx.x;
    unsigned int c_j = blockDim.y * blockIdx.y + threadIdx.y;
    /* 执行计算 */
    half output = 0.0;
    for (int k=0; k < K; ++k) {
        output += d_a[c_i * K + k] * d_b[k * N + c_j];
    }
    /* 结果写回 */
    d_c[c_i * N + c_j] = output;

}

template <const int tile>
__global__ void sgemm_tile(half* d_a, half* d_b, half* d_c, int M, int N, int K) {
    /* 当K过大时，一个block无法存入所有值，需要对K进行分块
       在grid上添加一维z来存储K/TILE个block，每个thread计算TILE个乘加
    */
    unsigned int c_i = blockDim.x * blockIdx.x + threadIdx.x;
    unsigned int c_j = blockDim.y * blockIdx.y + threadIdx.y;
    unsigned int tile_idx = blockIdx.z;

    half output = 0.0;
    size_t a_idx, b_idx;

    if (c_i < M && c_j < N) {
        #pragma unroll
        for (int k=0; k < tile; ++k) {
            a_idx = c_i * K + k + tile_idx * tile;
            b_idx = (k + tile_idx * tile) * N + c_j;
            if (a_idx < M*K && b_idx < K*N) {
                output += d_a[a_idx] * d_b[b_idx];
            }
        }
        atomicAdd(&d_c[c_i * N + c_j], output);
    }
}


__global__ void sgemm_tile_share(half* d_a, half* d_b, half* d_c, int M, int N, int K) {
    /* 在tile的基础上，增加共享内存 */
    __shared__ half sdata_a[TILE][TILE];
    __shared__ half sdata_b[TILE][TILE];

    unsigned int c_i = blockDim.x * blockIdx.x + threadIdx.x;
    unsigned int c_j = blockDim.y * blockIdx.y + threadIdx.y;
    unsigned int tile_idx = blockIdx.z;
    
    unsigned int tidx_y = threadIdx.y;
    unsigned int tidx_x = threadIdx.x;

    half output = 0.0;
    size_t a_idx, b_idx;
    /* 读取数据到共享内存中 */
    for(int i=0; i < TILE; ++i) {
        a_idx = c_i * K + i + TILE * tile_idx;
        b_idx = (i + TILE * tile_idx) * N + c_j;
        sdata_a[tidx_y][tidx_x] = d_a[a_idx];
        sdata_b[tidx_x][tidx_y] = d_b[b_idx];
        __syncthreads();
    } 

    /* 从共享内存读取数据完成计算 */
    for (int k=0; k < TILE; ++k) {
        output += sdata_a[tidx_y][k] * sdata_b[tidx_x][k];
    }
    atomicAdd(&d_c[c_i * N + c_j], output);
}


__global__ void gemm_share(half* d_a, half* d_b, half* d_c, int N, int K, int M) {
    /* 在baseline的基础上添加共享内存来存放输入值
        TODO:待完成，目前的线程组织分配还有问题
    */
    __shared__ half sdata_a[SHARED_BLOCK_SIZE][SHARED_BLOCK_SIZE];
    __shared__ half sdata_b[SHARED_BLOCK_SIZE][SHARED_BLOCK_SIZE];

    unsigned int c_i = blockDim.x * blockIdx.x + threadIdx.x;
    unsigned int c_j = blockDim.y * blockIdx.y + threadIdx.y;

    /* 拷贝数据到共享内存,将数据拆分成K/SHARED_BLOCK_SIZE(此处为方便写代码假设K=M=N) */
    unsigned int sblocks = (K + SHARED_BLOCK_SIZE -1 ) / SHARED_BLOCK_SIZE;
    for (int m=0; m < sblocks; ++m) {
        sdata_a[threadIdx.y][threadIdx.x] = d_a[(c_i + m * SHARED_BLOCK_SIZE + threadIdx.y) * K + m * SHARED_BLOCK_SIZE + threadIdx.x];
        sdata_b[threadIdx.y][threadIdx.x] = d_b[(m * SHARED_BLOCK_SIZE + threadIdx.x) * N + c_j + m * SHARED_BLOCK_SIZE + threadIdx.y];
        __syncthreads();
    }
    /* 计算矩阵乘法 */
    half output = 0.0;
    for (int i=0; i < SHARED_BLOCK_SIZE; ++i) {
        output += sdata_a[threadIdx.y][threadIdx.x] * sdata_b[threadIdx.y][threadIdx.x];
    }
    /* 写回global mem */
    d_c[c_i * N + c_j] += output;
    
}

template<
    const int BLOCK_SIZE_M,     // 每个线程块计算的矩阵C的连续行的数量
    const int BLOCK_SIZE_K,     // 每个线程块加载到共享内存中的矩阵A的连续列的数量
    const int BLOCK_SIZE_N,     // 每个线程块计算的矩阵C的连续列的数量
    const int THREAD_SIZE_Y,    // 每个线程计算矩阵C的block的行数
    const int THREAD_SIZE_X,    // 每个线程计算矩阵C的block的列数
    const bool ENABLE_DOUBLE_BUFFER //是否启用数据预取
    >
__global__ void gemm_without_prefectch(half* __restrict__ d_a, half* __restrict__ d_b, half* __restrict__ d_c, int M, int K, int N) {
    /*没有采用数据预取的矩阵乘法函数
    线程组织：
        设置grid：M / BLOCK_SIZE_M, block: BLOCK_SZIE_M / THREAD_SIZE_Y
        假如输入M=2048，N=2048，那么grid= 16 * 16,block=16 * 16
    每个block的计算逻辑：
        一共需要256次迭代，每次迭代：
            1.将矩阵A里面的128x8个元素和矩阵B里面的8x128个元素存入共享内存中。
            2.然后这个block中的256个线程把结果计算出来。每个线程的计算逻辑：
                2.1 每个线程需要进行8次迭代（BLOCK_SIZE_K）
                2.2 每次迭代中，每个线程从共享内存拿到A矩阵的一小列（8个数）和B矩阵的一小行（8个数）
                2.3 线程将这8+8个元素存入寄存器中。
                2.4 每个线程负责8x8=64个元素计算，一共会有64个FFMA指令
            3.计算完成后进入下一次迭代
        

    */
}

template<
    const int BLOCK_SIZE_M,     // 每个线程块计算的矩阵C的连续行的数量
    const int BLOCK_SIZE_K,     // 每个线程块加载到共享内存中的矩阵A的连续列的数量
    const int BLOCK_SIZE_N,     // 每个线程块计算的矩阵C的连续列的数量
    const int THREAD_SIZE_Y,    // 每个线程计算矩阵C的block的行数
    const int THREAD_SIZE_X,    // 每个线程计算矩阵C的block的列数
    const bool ENABLE_DOUBLE_BUFFER //是否启用数据预取
    >
__global__ void gemm_prefetch(half* __restrict__ d_a, half* __restrict__ d_b, half* __restrict__ d_c, int M, int K, int N) {
    /*采用数据预取的矩阵乘法函数，与上面函数的区别主要是两个方面：
        1. 开启的共享内存和寄存器数量：需要开启两倍的共享内存和寄存器数量。共享内存：bmxbk->bmxbkx2, bkxbn->bkxbnx2，寄存器：rm+rn->rmx2+rnx2
        2. 提前将一些数据放置在共享内存中
    每个block的计算逻辑：
        1. 先将第0轮迭代的数据存入共享内存，将线程第0次迭代的数据存入寄存器
        2. 执行第i次迭代，读取第i-1次存入的共享内存数据，线程第i此迭代读取第i-1次迭代的数据；同时存入第i+1此迭代的共享内存数据和线程第i+1次迭代的寄存器数据
        3. 通过多个FFMA计算来掩盖从global mem读取数据产生的延迟（需要有足够多的FFMA才能掩盖，所以这里设置了64个计算）
    */
    
    /* block idx */
    unsigned int bx = blockIdx.x;
    unsigned int by = blockIdx.y;

    /* thread idx */
    unsigned int tx = threadIdx.x;
    unsigned int ty = threadIdx.y;

    /* 每个block有多少个x方向和y方向的线程 */
    const int THREAD_X_PER_BLOCK = BLOCK_SIZE_N / THREAD_SIZE_X;
    const int THREAD_Y_PER_BLOCK = BLOCK_SIZE_M / THREAD_SIZE_Y;
    const int THREAD_PER_BLOCK = THREAD_X_PER_BLOCK * THREAD_Y_PER_BLOCK;

    /* 当前线程在此block中的id号 */
    const int tid = ty * THREAD_X_PER_BLOCK + tx;

    /* 设置共享内存和寄存器 */
    __shared__ half As[2][BLOCK_SIZE_K][BLOCK_SIZE_M];     // 为加快后续的访存，进行了一次转置，为了预取，开了两倍的buffer，一半用来read数据，一半用来write数据
    __shared__ half Bs[2][BLOCK_SIZE_K][BLOCK_SIZE_N];     //预取，开了两倍的buffer，一半用来read数据，一半用来write数据

    /* 用来临时存储矩阵C的计算结果 */
    half accum[THREAD_SIZE_Y][THREAD_SIZE_X] = {0};
    /* 矩阵A的寄存器存储，直接申请一个小数组即可，编译时编译器会将其放置在寄存器中，如果数组较大，会溢出到本地内存中，这时访问数组的时钟周期会达到几百 */
    half frag_a[2][THREAD_SIZE_Y];
    /* 矩阵B的寄存器存储 */
    half frag_b[2][THREAD_SIZE_X];

    /* (global->shared)加载数据次数：加载矩阵A的一个共享内存块，使用half4从主存读取数据，需要加载的次数 */
    const int ldg_num_a = BLOCK_SIZE_M * BLOCK_SIZE_K / (THREAD_PER_BLOCK * 2);
    const int ldg_num_b = BLOCK_SIZE_K * BLOCK_SIZE_N / (THREAD_PER_BLOCK * 2);
    /* (global->shared)将数据存入共享内存需要调用寄存器数量 */
    half ldg_a_reg[2 * ldg_num_a];
    half ldg_b_reg[2 * ldg_num_b];

    /* 每四个数进行一次访存操作，计算每行和每列需要进行多少次访存 */
    const int A_TILE_THREAD_PER_ROW = BLOCK_SIZE_K / 2;
    const int B_TILE_THREAD_PER_ROW = BLOCK_SIZE_N / 2;

    const int A_TILE_THREAD_PER_COL = BLOCK_SIZE_M / 2;
    const int B_TILE_THREAD_PER_COL = BLOCK_SIZE_K / 2;

    /* 计算当前进程负责的数据访存（x, y）->(此block mem中起始点的坐标) */
    const int A_TILE_ROW_START = tid / A_TILE_THREAD_PER_ROW;
    const int B_TILE_ROW_START = tid / B_TILE_THREAD_PER_ROW;

    const int A_TILE_COL_START = tid % A_TILE_THREAD_PER_ROW * 2;
    const int B_TILE_COL_START = tid % B_TILE_THREAD_PER_ROW * 2;

    /* 计算每个线程需要加载多少次数据 */
    const int A_TILE_ROW_STRIDE = THREAD_PER_BLOCK / A_TILE_THREAD_PER_ROW;
    const int B_TILE_ROW_STRIDE = THREAD_PER_BLOCK / B_TILE_THREAD_PER_ROW;

    d_a = &d_a[(BLOCK_SIZE_M * by) * K];    // 当前block负责的A的首地址；
    d_b = &d_b[BLOCK_SIZE_N * bx];          // 当前block负责的B的首地址；

    /* 加载矩阵A到共享内存：数据流动方向global mem -> register -> shared mem */
    #pragma unroll
    for (int i=0; i < BLOCK_SIZE_M; i += A_TILE_ROW_STRIDE) {
        int ldg_index = i / A_TILE_ROW_STRIDE * 2;
        FETCH_HALF2(ldg_a_reg[ldg_index]) = FETCH_HALF2(d_a[OFFSET(A_TILE_ROW_START + i,    // row
                                                                 A_TILE_COL_START,                              //col
                                                                 K)]);
        As[0][A_TILE_COL_START][A_TILE_ROW_START + i] = ldg_a_reg[ldg_index];
        As[0][A_TILE_COL_START+1][A_TILE_ROW_START + i] = ldg_a_reg[ldg_index+1];
    }
    /* 加载矩阵B到共享内存 */
    #pragma unroll
    for (int i=0; i < BLOCK_SIZE_K; i += B_TILE_ROW_STRIDE) {
        FETCH_HALF2(Bs[0][B_TILE_ROW_START + i][B_TILE_COL_START]) = FETCH_HALF2(d_b[OFFSET(
                                                                                            B_TILE_ROW_START + i,                   // row
                                                                                            B_TILE_COL_START,   // col
                                                                                            N
                                                                                    )]);
    }
    __syncthreads();

    /* 将共享内存数据预取到寄出器中 */
    #pragma unroll
    for (int thread_y = 0; thread_y < THREAD_SIZE_Y; thread_y += 2) {
        FETCH_HALF2(frag_a[0][thread_y]) = FETCH_HALF2(As[0][0][THREAD_SIZE_Y * ty + thread_y]);
    }

    #pragma unroll
    for (int thread_x = 0; thread_x < THREAD_SIZE_X; thread_x += 2) {
        FETCH_HALF2(frag_b[0][thread_x]) = FETCH_HALF2(Bs[0][0][THREAD_SIZE_X * tx + thread_x]);
    }

    /* 执行迭代：包括外层的256个迭代和每个线程的迭代 */
    int write_stage_idx = 1;
    int tile_idx = 0;
    do{
        tile_idx += BLOCK_SIZE_K;
        /* 如果还有下一个迭代，加载下一个block的数据到寄存器 */
        if (tile_idx < K) {
            #pragma unroll
            for (int i=0; i < BLOCK_SIZE_M; i += A_TILE_ROW_STRIDE) {
                int ldg_index = i / A_TILE_ROW_STRIDE * 2;
                FETCH_HALF2(ldg_a_reg[ldg_index]) = FETCH_HALF2(d_a[OFFSET(
                                                                            A_TILE_ROW_START + i,
                                                                            A_TILE_COL_START + tile_idx,
                                                                            K
                                                                        )]);
            }
            #pragma unroll
            for (int i=0; i < BLOCK_SIZE_K; i += B_TILE_ROW_STRIDE) {
                int ldg_index = i / B_TILE_ROW_STRIDE * 2;
                FETCH_HALF2(ldg_b_reg[ldg_index]) = FETCH_HALF2(d_b[OFFSET(
                                                                            tile_idx + B_TILE_ROW_START + i,
                                                                            B_TILE_COL_START,
                                                                            N                                                        
                                                                        )]);
            }
        }
        /* 该变量表示需要从As的哪个空间进行读数 */
        int load_stage_idx = write_stage_idx ^ 1;
        /* 需要完成7次小迭代，由于在小迭代中也采用了双缓冲的方式，需要将下一轮的小迭代数据提前写入寄存器中 */
        #pragma unroll
        for (int j = 0; j < BLOCK_SIZE_K-1; ++j) {
            /* 从共享内存As读取数据到寄存器frag_a中 */
            #pragma unroll
            for (int thread_y=0; thread_y < THREAD_SIZE_Y; thread_y += 2) {
                FETCH_HALF2(frag_a[(j+1)%2][thread_y]) = FETCH_HALF2(As[load_stage_idx][j+1][THREAD_SIZE_Y * ty + thread_y]);
            }
            /* 从共享内存Bs读取数据到寄存器frag_b中 */
            #pragma unroll
            for (int thread_x=0; thread_x < THREAD_SIZE_X; thread_x +=2) {
                FETCH_HALF2(frag_b[(j+1)%2][thread_x]) = FETCH_HALF2(Bs[load_stage_idx][j+1][THREAD_SIZE_X * tx + thread_x]);
            }

            /* 计算矩阵C的值，并将其存入accum寄存器中 */
            #pragma unroll
            for (int thread_y = 0; thread_y < THREAD_SIZE_Y; ++thread_y) {
                #pragma unroll
                for (int thread_x=0; thread_x < THREAD_SIZE_X; ++thread_x) {
                    accum[thread_y][thread_x] += frag_a[j%2][thread_y] * frag_b[j%2][thread_x];
                }
            }
        }

        /* 将存储在临时寄存器的数据搬运到共享内存中 */
        if (tile_idx < K) {
            /* 将A的数据写回共享内存 */
            #pragma unroll
            for (int i=0; i < BLOCK_SIZE_M; i += A_TILE_ROW_STRIDE) {
                int ldg_index = i / A_TILE_ROW_STRIDE * 2;
                As[write_stage_idx][A_TILE_COL_START][A_TILE_ROW_START+i] = ldg_a_reg[ldg_index];
                As[write_stage_idx][A_TILE_COL_START+1][A_TILE_ROW_START+i] = ldg_a_reg[ldg_index+1];
            }
            /*将B的数据写回共享内存*/
            #pragma unroll
            for (int i=0; i < BLOCK_SIZE_K; i += B_TILE_ROW_STRIDE) {
                int ldg_index = i / B_TILE_ROW_STRIDE * 2;
                FETCH_HALF2(Bs[write_stage_idx][B_TILE_ROW_START + i][B_TILE_COL_START]) = FETCH_HALF2(ldg_b_reg[ldg_index]);
            }
            /* 使用double buffer，只需要一个sync */
            __syncthreads();

            /* 切换状态 */
            write_stage_idx ^= 1;
        }

        /* 完成最后一个小迭代以及寄存器的预取 */
        #pragma unroll
        for (int thread_y = 0; thread_y < THREAD_SIZE_Y; thread_y += 2) {
            FETCH_HALF2(frag_a[0][thread_y]) = FETCH_HALF2(As[load_stage_idx^1][0][THREAD_SIZE_Y * ty + thread_y]);
        }
        /* 从共享内存加载矩阵B的数据 */
        #pragma unroll
        for (int thread_x = 0; thread_x < THREAD_SIZE_X; thread_x += 2) {
            FETCH_HALF2(frag_b[0][thread_x]) = FETCH_HALF2(Bs[load_stage_idx^1][0][THREAD_SIZE_X * tx + thread_x]);
        }

        /* 计算最后一个tile的值 */
        #pragma unroll
        for (int thread_y=0; thread_y < THREAD_SIZE_Y; ++thread_y) {
            #pragma unroll
            for (int thread_x=0; thread_x <THREAD_SIZE_X; ++thread_x) {
                accum[thread_y][thread_x] += frag_a[1][thread_y] * frag_b[1][thread_x];
            }
        }
    }while(tile_idx < K);

    /* 将计算结果写回主存 */
    #pragma unroll
    for (int thread_y=0; thread_y < THREAD_SIZE_Y; ++thread_y) {
        #pragma unroll
        for (int thread_x=0; thread_x < THREAD_SIZE_X; thread_x += 2) {
            FETCH_HALF2(d_c[OFFSET(
                BLOCK_SIZE_M * by + ty * THREAD_SIZE_Y + thread_y,
                BLOCK_SIZE_N * bx + tx * THREAD_SIZE_X + thread_x,
                N
            )]) = FETCH_HALF2(accum[thread_y][thread_x]);
        }
    }
}


__global__ void hgemm_tensorcore(half *d_a, half *d_b, half *d_c, int N, int K, int M) {
    /*混合精度计算，使用tensorcore完成矩阵计算*/
    int k_tile = div_ceil(K, WMMA_K);

    int idx_row = blockIdx.y * WMMA_M;
    int idx_col = blockIdx.x * WMMA_N;

    if (idx_row < M && idx_col < N) {
        wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, half> frag_c;
        /*frag_c赋值*/
        wmma::fill_fragment(frag_c, 0.0f);
        /*frag_c需要从主存读数赋值*/
        wmma::load_matrix_sync(frag_c, d_c + idx_row * N + idx_col, N, wmma::mem_row_major);
        for (int i=0; i < k_tile; ++i) {
            /**/
            wmma::fragment<wmma::matrix_a, WMMA_M, WMMA_N, WMMA_K, half, wmma::row_major> frag_a;
            wmma::fragment<wmma::matrix_b, WMMA_M, WMMA_N, WMMA_K, half, wmma::row_major> frag_b;

            /*从global mem拷贝数据*/
            wmma::load_matrix_sync(frag_a, d_a + idx_row * K + i * WMMA_K, K);
            wmma::load_matrix_sync(frag_b, d_b + (i * WMMA_K) * N + idx_col, N);

            /*计算*/
            wmma::mma_sync(frag_c, frag_a, frag_b, frag_c);
        }

        wmma::store_matrix_sync(d_c + idx_row * N + idx_col, frag_c, N, wmma::mem_row_major);
    }

}



void cublas_gemm(half* d_a, half* d_b, half* d_c, int N, int K, int M) {
    /* cublas版本单精度矩阵乘法实现 */
    
    hipblasHandle_t blas_handle;
    hipblasCreate(&blas_handle);
    half alpha = 1.0f;
    half beta = 0.0f;
    hipblasHgemm(blas_handle, HIPBLAS_OP_N, HIPBLAS_OP_T, 
                M, N, K, &alpha,
                d_a, M, d_b, K, &beta, d_c, M 
                );
}

void hgemm_cublas_launcher(half* a, half* b, half* c, int N, int K, int M) {
    /* 分配GPU资源 */
    int sizeA = M*K;
    int sizeB = K*N;
    int sizeC = M*N;
    half* d_a;
    half* d_b;
    half* d_c;
    CUDA_CHECK(hipMalloc(&d_a, sizeA * sizeof(half)));
    CUDA_CHECK(hipMalloc(&d_b, sizeB * sizeof(half)));
    CUDA_CHECK(hipMalloc(&d_c, sizeC * sizeof(half)));

    /* 拷贝数据到global mem */
    CUDA_CHECK(hipMemcpy(d_a, a, sizeA * sizeof(half), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_b, b, sizeB * sizeof(half), hipMemcpyHostToDevice));

    /* cublas实现 */
    cublas_gemm(d_a, d_b, d_c, N, K, M);

    /* 结果拷回host内存 */
    CUDA_CHECK(hipMemcpy(c, d_c, sizeC * sizeof(half), hipMemcpyDeviceToHost));
    CUDA_CHECK(hipDeviceSynchronize());
    LAST_KERNEL_CHECK();

    /* 释放显存 */
    CUDA_CHECK(hipFree(d_c));
    CUDA_CHECK(hipFree(d_b));
    CUDA_CHECK(hipFree(d_a));

}

void hgemm_kernel_launcher(half* a, half* b, half* c, int N, int K, int M) {
    /* 分配GPU资源 */
    int sizeA = M*K;
    int sizeB = K*N;
    int sizeC = M*N;
    half* d_a;
    half* d_b;
    half* d_c;
    CUDA_CHECK(hipMalloc(&d_a, sizeA * sizeof(half)));
    CUDA_CHECK(hipMalloc(&d_b, sizeB * sizeof(half)));
    CUDA_CHECK(hipMalloc(&d_c, sizeC * sizeof(half)));

    /* 拷贝数据到global mem */
    CUDA_CHECK(hipMemcpy(d_a, a, sizeA * sizeof(half), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_b, b, sizeB * sizeof(half), hipMemcpyHostToDevice));


    /* baseline：naive版本的gemm，每个线程处理一个output元素 */
    // dim3 block(BLOCK_SIZE, BLOCK_SIZE);
    // dim3 grid((M + BLOCK_SIZE - 1)/ BLOCK_SIZE, (N + BLOCK_SIZE - 1) / BLOCK_SIZE);
    // gemm_baseline<<<grid, block>>>(d_a, d_b, d_c, N, K, M);

    /*优化一：对K使用tile*/
    // dim3 block_1(BLOCK_SIZE, BLOCK_SIZE);
    // dim3 grid_1((M + BLOCK_SIZE - 1)/ BLOCK_SIZE, (N + BLOCK_SIZE - 1) / BLOCK_SIZE, (K + TILE - 1) / TILE);
    // sgemm_tile<TILE><<<grid_1, block_1>>>(d_a, d_b, d_c, M, N, K);

    // /* 优化二：tileK + shared mem */
    // dim3 block_2(BLOCK_SIZE, BLOCK_SIZE);
    // dim3 grid_2((M + BLOCK_SIZE - 1)/ BLOCK_SIZE, (N + BLOCK_SIZE - 1) / BLOCK_SIZE, (K + TILE - 1) / TILE);
    // sgemm_tile_share<<<grid_2, block_2>>>(d_a, d_b, d_c, M, N, K);
    /* 优化二：使用共享内存+tile */
    // gemm_share<<<grid, block>>>(d_a, d_b, d_c, N, K, M);

    // /* 优化三：使用tile+prefetch策略 */
    // const int BLOCK_SIZE_M = 128;     // 每个线程块计算的矩阵C的连续行的数量
    // const int BLOCK_SIZE_K = 8;     // 每个线程块加载到共享内存中的矩阵A的连续列的数量
    // const int BLOCK_SIZE_N = 128;     // 每个线程块计算的矩阵C的连续列的数量
    // const int THREAD_SIZE_Y = 8;    // 每个线程计算矩阵C的block的行数
    // const int THREAD_SIZE_X = 8;    // 每个线程计算矩阵C的block的列数
    // const bool ENABLE_DOUBLE_BUFFER = false;

    // dim3 block_size(BLOCK_SIZE_N / THREAD_SIZE_X, BLOCK_SIZE_M / THREAD_SIZE_Y);
    // dim3 grid_size(N / BLOCK_SIZE_N, M / BLOCK_SIZE_M);
    
    // gemm_prefetch<BLOCK_SIZE_M, BLOCK_SIZE_K, BLOCK_SIZE_N, THREAD_SIZE_Y, THREAD_SIZE_X, ENABLE_DOUBLE_BUFFER>
    // <<<grid_size, block_size>>>(d_a, d_b, d_c, M, K, N);


    /*tensor core实现*/

    dim3 block_h(WARP_SIZE);
    dim3 grid_h(div_ceil(M, WMMA_M), div_ceil(N, WMMA_N));
    hgemm_tensorcore<<<grid_h, block_h>>>(d_a, d_b, d_c, M, K, N);


    /* 结果拷回host内存 */
    CUDA_CHECK(hipMemcpy(c, d_c, sizeC * sizeof(half), hipMemcpyDeviceToHost));
    CUDA_CHECK(hipDeviceSynchronize());
    LAST_KERNEL_CHECK();

    /* 释放显存 */
    CUDA_CHECK(hipFree(d_c));
    CUDA_CHECK(hipFree(d_b));
    CUDA_CHECK(hipFree(d_a));
}